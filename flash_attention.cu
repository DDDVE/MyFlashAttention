#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>

#define SPEED_FACTOR (1000000000000)

float get_flops_speed(int B, int H, int N, int d, float time, bool causal=false, int mode=0)
{
    /* 0=fwd 1=bwd 2=both */
    int flops = 4 * B * N * N * H * d;
    if (causal)
    {
        flops /= 2;
    }
    if (mode == 1)
    {
        flops *= 2.5;
    }
    if (mode == 2)
    {
        flops *= 3.5;
    }
    return flops / time / SPEED_FACTOR;
}

__global__ void initToInfinity_float(float* m, int N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
    {
        return;
    }
    m[idx] = -INFINITY;
}

/* 读取二维数据 */
__device__ __forceinline__ float read_2d(const float* array, int row, int col, int cols_per_row)
{
    return array[row * cols_per_row + col];
}

/* 写入二维数据 */
__device__ __forceinline__ void write_2d(float* array, int row, int col, int cols_per_row, float& data)
{
    array[row * cols_per_row + col] = data;
}

/* 读取四维数据 */
__device__ __forceinline__ float read_4d(const float* array, int batch, int head, int row, int col, int heads_per_batch, int rows_per_head, int cols_per_row)
{
    return array[batch * (heads_per_batch * rows_per_head * cols_per_row) + 
        head * (rows_per_head * cols_per_row) + 
        row * cols_per_row + 
        col];
}

/* 写入四维数据 */
__device__ __forceinline__ void write_4d(float* array, int batch, int head, int row, int col, 
int heads_per_batch, int rows_per_head, int cols_per_row, float& data)
{
    array[batch * (heads_per_batch * rows_per_head * cols_per_row) + 
        head * (rows_per_head * cols_per_row) + 
        row * cols_per_row + 
        col] = data;
}

/* warp内部归约求和 */
__inline__ __device__ float warpReduceSum(float val, int Bc) {
    for (int offset = Bc / 2; offset > 0; offset /= 2) {  
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;  // 线程 0 存储最终和
}

/* warp内部归约求最大值 */
__device__ float warpReduceMax(float val, int Bc) {
    // 循环进行归约，offset 每次减半，直到 offset=0
    for (int offset = Bc / 2; offset > 0; offset /= 2) {
        // 读取当前线程向右偏移 offset 线程的值
        float tmp = __shfl_down_sync(0xffffffff, val, offset);
        val = fmaxf(val, tmp);  // 取两个线程的最大值
    }
    return val;
}

/* flash attention核函数, 每个线程块处理一个Br Bc分块 */
__global__ void flash_attention(float* __restrict__ Q, float* __restrict__ K, float* __restrict__ V, 
                                float* __restrict__ O, float* __restrict__ l, float* __restrict__ m,
                            int B, int H, int N, int d, int Br, int Bc, int Tr, int Tc)
{
    /* 获取当前线程位置 */
    // 得出当前线程位于哪个B和H
    int batch_id = blockIdx.x;
    int head_id = blockIdx.y;
    // 线程对应子矩阵中的行号列号
    int local_col_id = threadIdx.x;
    int local_row_id = threadIdx.y;
    /* 划分共享内存 */
    extern __shared__ __align__(16) float shared_mem[];
    // 划分Qi
    float* shared_Qi = shared_mem;                                      // Br * d
    // 划分Ki
    float* shared_Ki = shared_Qi + ((Br * d + 31) / 32 * 32);           // Bc * d
    // 划分Vi
    float* shared_Vi = shared_Ki + ((Bc * d + 31) / 32 * 32);           // Bc * d
    // 划分Oi
    float* shared_Oi = shared_Vi + ((Bc * d + 31) / 32 * 32);           // Br * d
    // 划分mi
    float* shared_mi = shared_Oi + ((Br * d + 31) / 32 * 32);           // Br
    // 划分li
    float* shared_li = shared_mi + ((Br * 1 + 31) / 32 * 32);           // Br
    // 划分Sij
    float* shared_Sij = shared_li + ((Br * 1 + 31) / 32 * 32);          // 存储当前块负责的局部QK^t, Br * Bc
    // 划分mij
    float* shared_mij = shared_Sij + ((Br * Bc + 31) / 32 * 32);        // 存储每一次Sij的行最大值, Br
    // 划分lij
    float* shared_lij = shared_mij + ((Br * 1 + 31) / 32 * 32);        // Br
    // 划分minew
    float* shared_mi_new = shared_lij + ((Br * 1 + 31) / 32 * 32);      // Br
    // 划分linew
    float* shared_li_new = shared_mi_new + ((Br * 1 + 31) / 32 * 32);   // Br

    // 向量化加载时的末尾索引
    int float4_end = d / 4 * 4;
    int float4_num = d / 4;

    float tmp = 0;
    /* 遍历所有分块 */ 
    for (int j = 0; j < Tc; j++)
    {
        // 全局列坐标
        int global_col_id = j * Bc + local_col_id;
        bool mask = global_col_id >= N;
        if (mask)
        {
            return;
        }
        // 实际的Bc
        int actual_Bc = (j == Tc - 1) ? (N % Bc == 0 ? Bc : N % Bc) : Bc;
        // 加载KV
        // 向量化加载
        for (int col = local_row_id; col < float4_num; col += Br)
        {
            int global_offset = batch_id * (H * N * float4_num) + head_id * (N * float4_num) + global_col_id * float4_num + col;
            float4 tmp = ((float4*)K)[global_offset];
            
            // 加载K
            *(float4*)(&shared_Ki[local_col_id * d + col*4]) = tmp;
            // 加载V
            tmp = ((float4*)V)[global_offset];
            *(float4*)(&shared_Vi[local_col_id * d + col*4]) = tmp;
        }
        // 逐元素加载剩余部分
        // for (int col = float4_num * 4; col < d; col += Br)
        // {
        //     // 加载K
        //     tmp = read_4d(K, batch_id, head_id, global_col_id, col, H, N, d);
        //     write_2d(shared_Ki, local_col_id, col, d, tmp);
        //     // 加载V
        //     tmp = read_4d(V, batch_id, head_id, global_col_id, col, H, N, d);
        //     write_2d(shared_Vi, local_col_id, col, d, tmp);
        // }

        for (int i = 0; i < Tr; i++)
        {
            //__syncthreads();
            // 全局行坐标
            int global_row_id = i * Br + local_row_id;
            // 判断是否掩盖当前线程
            bool mask = (global_row_id >= N);
            if (mask)
            {
                continue;
            }
            // 实际的Br
            // int actual_Br = (i == Tr - 1) ? (N % Br == 0 ? Br : N % Br) : Br;

            /* 从全局内存中加载数据 */
            /* 加载Q O */
            // 向量化加载
            for (int col = local_col_id; col < float4_num; col += actual_Bc)
            {
                int global_offset = batch_id * (H * N * float4_num) + head_id * (N * float4_num) + global_row_id * float4_num + col;
                float4 tmp = ((float4*)Q)[global_offset];
                *(float4*)(&shared_Qi[local_row_id * d + col*4]) = tmp;

                tmp = ((float4*)O)[global_offset];
                *(float4*)(&shared_Oi[local_row_id * d + col*4]) = tmp;
            }
            // 逐元素加载剩余部分
            // for (int col = float4_num*4; col < d; col += actual_Bc)
            // {
            //     // 加载Q
            //     tmp = read_4d(Q, batch_id, head_id, global_row_id, col, H, N, d);
            //     write_2d(shared_Qi, local_row_id, col, d, tmp);
            //     // 加载O
            //     tmp = read_4d(O, batch_id, head_id, global_row_id, col, H, N, d);
            //     write_2d(shared_Oi, local_row_id, col, d, tmp);
            // }

            // 加载li mi
            if (local_col_id == 0)
            {
                tmp = read_4d(l, batch_id, head_id, global_row_id, 0, H, N, 1);
                write_2d(shared_li, local_row_id, 0, 1, tmp);
                tmp = read_4d(m, batch_id, head_id, global_row_id, 0, H, N, 1);
                write_2d(shared_mi, local_row_id, 0, 1, tmp);
            }
            __syncthreads();

            /* 开始计算 */
            // 计算Sij = Qi * Ki^t
            tmp = 0;
            // 先处理能被4整除的部分
            for (int col = 0; col < float4_end; col += 4)
            {
                float4 q_val = *((float4*)&shared_Qi[local_row_id * d + col]);
                float4 k_val = *((float4*)&shared_Ki[local_col_id * d + col]);
                tmp += q_val.x * k_val.x + q_val.y * k_val.y + q_val.z * k_val.z + q_val.w * k_val.w;
            }
            // 处理剩余不满4的部分
            // for (int i = float4_end; i < d; i++)
            // {
            //     tmp += (read_2d(shared_Qi, local_row_id, i, d) * read_2d(shared_Ki, local_col_id, i, d));
            // }
            // 将结果写到Sij的[local_row_id, local_col_id]处, 无需原子因为一个线程只处理Sij的一个位置
            write_2d(shared_Sij, local_row_id, local_col_id, Bc, tmp);
            __syncthreads();

            // 计算mij = rowmax(Sij), 
            tmp = read_2d(shared_Sij, local_row_id, local_col_id, Bc);
            float row_max = warpReduceMax(tmp, actual_Bc);
            if (local_col_id == 0)
            {
                write_2d(shared_mij, local_row_id, 0, 1, row_max);
                // 更新minew
                tmp = read_2d(shared_mi, local_row_id, 0, 1);
                tmp = fmaxf(tmp, row_max);
                write_2d(shared_mi_new, local_row_id, 0, 1, tmp);
                // lij清零
                tmp = 0;
                write_2d(shared_lij, local_row_id, 0, 1, tmp);
            }
            __syncthreads();
            // 计算Pij = exp(Sij - mij)
            // 读取Sij和mij
            tmp = read_2d(shared_Sij, local_row_id, local_col_id, Bc);
            // 计算exp(平移)
            tmp = expf(tmp - read_2d(shared_mij, local_row_id, 0, 1));
            // 直接写回Sij
            write_2d(shared_Sij, local_row_id, local_col_id, Bc, tmp);

            // 计算rowsum
            float warpsum = warpReduceSum(tmp, actual_Bc);
            if (local_col_id % warpSize == 0)
            {
                atomicAdd(&(shared_lij[local_row_id]), warpsum);
            }
            __syncthreads();

            // 计算linew = exp(mi - minew) * li + exp(mij - minew) * lij
            if (local_col_id == 0)
            {
                float mi = read_2d(shared_mi, local_row_id, 0, 1);
                float minew = read_2d(shared_mi_new, local_row_id, 0, 1);
                float mij = read_2d(shared_mij, local_row_id, 0, 1);
                float li = read_2d(shared_li, local_row_id, 0, 1);
                float lij = read_2d(shared_lij, local_row_id, 0, 1);
                tmp = expf(mi - minew) * li + expf(mij - minew) * lij;
                write_2d(shared_li_new, local_row_id, 0, 1, tmp);
            }
            __syncthreads();

            /* 计算diag(lnew)^-1 * (diag(li)*exp(mi-minew)*Oi + exp(mij - minew)*Sij*Vj) */
            float mi = read_2d(shared_mi, local_row_id, 0, 1);
            float minew = read_2d(shared_mi_new, local_row_id, 0, 1);
            float mij = read_2d(shared_mij, local_row_id, 0, 1);
            float li = read_2d(shared_li, local_row_id, 0, 1);
            float linew = read_2d(shared_li_new, local_row_id, 0, 1);
            float exp_mij_minew = expf(mij - minew);

            // 乘以Oi
            for (int col = local_col_id; col < d; col += actual_Bc)
            {
                tmp = expf(mi - minew) * li;
                float Oi = read_2d(shared_Oi, local_row_id, col, d);
                tmp *= Oi;
                float tmp_1 = 0;
                for (int k = 0; k < actual_Bc; k++) 
                {
                    tmp_1 += (read_2d(shared_Sij, local_row_id, k, Bc) * read_2d(shared_Vi, k, col, d));
                }
                tmp_1 *= exp_mij_minew;
                tmp += tmp_1;
                tmp /= linew;
                write_2d(shared_Oi, local_row_id, col, d, tmp);
            }


            // 从共享内存写回全局内存
            // 写Oi 向量化
            for (int col = local_col_id; col < float4_num; col += actual_Bc)    
            {
                int global_offset = batch_id * (H * N * float4_num) + head_id * (N * float4_num) + global_row_id * float4_num + col;
                float4 tmp = *(float4*)(&shared_Oi[local_row_id * d + col*4]);
                ((float4*)O)[global_offset] = tmp;
            }
            // 处理不整除的部分
            for (int col = float4_num * 4; col < d; col += actual_Bc)
            {
                float tmp = read_2d(shared_Oi, local_row_id, col, d);
                write_4d(O, batch_id, head_id, global_row_id, col, H, N, d, tmp);
            }
                
            if (local_col_id == 0)
            {
                // 写linew
                tmp = read_2d(shared_li_new, local_row_id, 0, 1);
                write_4d(l, batch_id, head_id, global_row_id, 0, H, N, 1, tmp);
                // 写minew
                tmp = read_2d(shared_mi_new, local_row_id, 0, 1);
                write_4d(m, batch_id, head_id, global_row_id, 0, H, N, 1, tmp);
            }
        }
    }
}

/* 主机函数 */
float flashAttentionHost(float* Q, float* K, float* V, float* O, int B, int H, int N, int d)
{
    printf("global K is [%p]\n", K);
    printf("global V is [%p]\n", V);
    /* 把共享内存上限设置为64KB */
    hipFuncSetAttribute(
        reinterpret_cast<const void*>(flash_attention),                      // 核函数名
        hipFuncAttributeMaxDynamicSharedMemorySize,
        65536                                 // 最大 64 KB
    );
    /* 获取SRAM大小M */
    hipDeviceProp_t device_prop;
    int device_id;
    // 获取设备id
    hipGetDevice(&device_id);
    std::cout << "Get device id = [" << device_id << "]" << std::endl;
    // 获取当前设备属性
    hipGetDeviceProperties(&device_prop, device_id);
    // 获取共享内存大小
    int M = device_prop.sharedMemPerBlock;
    std::cout << "Get shared mem size = [" << M << "]" << std::endl;
    /* 计算块大小 */
    int Br = 64, Bc = 16;
    // 调整 Br 和 Bc 的计算，确保共享内存不超标
    // int Bc = std::min(N, M / (16 * d));
    // int Br = std::min(d, M / (32 * Bc));

    // 如果计算出的 Br 或 Bc 太小，可以再进一步限制
    // Br = std::max(Br, 32); // Br 至少为 32
    // Bc = std::max(Bc, 32); // Bc 至少为 32

    std::cout << "Br is [" << Br << "], Bc is [" << Bc << "]" << std::endl;
    // 判断共享内存是否超出限制
    int shared_mem_size = 
                ((Br * d + 31) / 32 * 32) + //shared_Qi
                ((Bc * d + 31) / 32 * 32) + // shared_Ki
                ((Bc * d + 31) / 32 * 32) + // shared_Vi
                ((Br * d + 31) / 32 * 32) + // shared_Oi
                ((Br * 1 + 31) / 32 * 32) + // shared_mi
                ((Br * 1 + 31) / 32 * 32) + // shared_li
                ((Br * Bc + 31) / 32 * 32) + // shared_Sij
                ((Br * 1 + 31) / 32 * 32) + // shared_mij
                ((Br * 1 + 31) / 32 * 32) + // shared_lij
                ((Br * 1 + 31) / 32 * 32) + // shared_mi_new
                ((Br * 1 + 31) / 32 * 32) ; // shared_li_new
    shared_mem_size *= sizeof(float);
    std::cerr << "Sahred memory need [" << shared_mem_size << "], have [" << M << "]" << std::endl;
    if (shared_mem_size > M)
    {
        return -1;
    }

    /* 初始化事件 */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* 分配设备空间 */
    // 分配l l需要每个BH都独立, 所以需要分配B*H个
    float* device_l;
    hipMalloc(&device_l, B * H * N * sizeof(float));
    hipMemset(device_l, 0, B * H * N * sizeof(float));
    // 分配m m需要每个BH都独立, 所以需要分配B*H个
    float* device_m;
    hipMalloc(&device_m, B * H * N * sizeof(float));
    // 使用核函数设置为负无穷
    int thread_per_block = 1024;
    int block_num = (B * H * N + thread_per_block - 1) / thread_per_block;
    initToInfinity_float<<<block_num, thread_per_block>>>(device_m, B * H * N);
    hipDeviceSynchronize();

    /* 启动核函数 */
    // 计算有几个行块几个列块
    int Tr = (N + Br - 1) / Br;
    int Tc = (N + Bc - 1) / Bc;
    std::cout << "Tr = [" << Tr << "], Tc = [" << Tc << "]" << std::endl;
    // 设置线程块和线程网格的大小
    dim3 block_dim(Bc, Br, 1);
    dim3 grid_dim(B , H, 1);
    // 开始计时
    hipEventRecord(start, 0);
    // 启动核函数 指定共享内存大小
    flash_attention<<<grid_dim, block_dim, shared_mem_size>>>(Q, K, V, O, device_l, device_m, B, H, N, d, Br, Bc, Tr, Tc);
    // 停止计时
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // 检查是否有错误
    hipError_t err = hipGetLastError();
    float miliseconds = -1;
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    else
    {
        // 计算时间
        hipEventElapsedTime(&miliseconds, start, stop);
        float flops_speed = get_flops_speed(B, H, N, d, miliseconds / 1000);
        std::cout << "Kernel exec time [" << miliseconds << "] ms. Speed [" << flops_speed << "] TFLops/s" <<  std::endl;
    }
    // 清理事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 释放资源
    hipFree(device_l);
    hipFree(device_m);
    return miliseconds;
}

